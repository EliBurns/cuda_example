#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#define PI 3.14159265359


typedef struct 
{
    double x;
    double y;
    double z;
} Vector3D;


/// 3D dot product
__device__
double dot_prod3D(Vector3D *V1, Vector3D *V2)
{
    return V1->x*V2->x + V1->y*V2->y + V1->z*V2->z;
} 


/// window intersection scalar
/// assumes V is unit vector and returns true if scalar exists 
__device__
bool view_scalar(Vector3D * V, Vector3D * C, int R, double * t)
{
    double dot_VC = dot_prod3D(V, C);
    double dot_CC = dot_prod3D(C, C);

    if ((dot_VC*dot_VC + R*R - dot_CC) < 0)
        return false;

    *t = dot_VC - sqrt(dot_VC*dot_VC + R*R - dot_CC); 
    
    return true;
}


/// intersection of view and sphere
__device__
void intersection(Vector3D * I, Vector3D * V, double t)
{
    I->x = t*V->x;
    I->y = t*V->y;
    I->z = t*V->z;
}


/// unit normal vector of sphere
__device__
void unit_norm_sphere(Vector3D *I, Vector3D *C, Vector3D *N)
{
    Vector3D ImC;

    ImC.x = I->x - C->x;
    ImC.y = I->y - C->y;
    ImC.z = I->z - C->z;

    double dot_ImC = dot_prod3D(&ImC, &ImC);

    N->x = (ImC.x) / sqrt(dot_ImC);
    N->y = (ImC.y) / sqrt(dot_ImC);
    N->z = (ImC.z) / sqrt(dot_ImC);
}


/// create shadow ray and compute brightness
__device__
double brightness(Vector3D *I, Vector3D *L, Vector3D *N)
{
    Vector3D S;
    Vector3D LmI;

    // calculate shadow ray
    LmI.x = L->x - I->x;
    LmI.y = L->y - I->y;
    LmI.z = L->z - I->z;

    double dot_LmI = dot_prod3D(&LmI, &LmI);

    S.x = (LmI.x) / sqrt(dot_LmI);
    S.y = (LmI.y) / sqrt(dot_LmI);
    S.z = (LmI.z) / sqrt(dot_LmI);
    
    // return the max between 0 and S.N
    double dot_SN = dot_prod3D(&S, N);

    return (dot_SN > 0) ? dot_SN : 0.0;
}


// kernel function
__global__
void raytrace(double * grid, int grid_p, int n_rays)
{
	// use cuda's random number generator
	int i =  blockDim.x*blockIdx.x + threadIdx.x;

	hiprandState_t state;
  	hiprand_init(i, 0, 0, &state);

    int radius = 6;
    double W_y = 10; double W_max = 10;
    double window_scale = (((double)grid_p) / ((double)(2*W_max)));
     
    // set up light source and sphere center position
    Vector3D L;
    L.x = 4; L.y = 4; L.z = -1; 
    
    Vector3D C;
    C.x = 0; C.y = 12; C.z = 0;

    Vector3D W; Vector3D V; Vector3D I; Vector3D N;

    for (int i = 0; i < n_rays; ++i)
    {
        double t, theta, phi, b;

        do
        {
   	    phi = (double) hiprand_uniform(&state) * (double) M_PI;
      	    theta = (double) hiprand_uniform(&state)  * (double) M_PI;
 
            V.x  = sin(theta) * cos(phi);
            V.y  = sin(theta) * sin(phi);
            V.z  = cos(theta);

            W.x = (W_y / V.y) * V.x;
            W.y = (W_y / V.y) * V.y; 
            W.z = (W_y / V.y) * V.z;

        } while ((!view_scalar(&V, &C, radius, &t)) || (fabs(W.x) > W_max) || (fabs(W.z) > W_max));

        intersection(&I, &V, t);

        unit_norm_sphere(&I, &C, &N);
        
        b = brightness(&I, &L, &N);        
        
        double x = (W.x + (double)W_max);
        double z = (W.z + (double)W_max);
        x = x * window_scale;
        z = z * window_scale;
		
        grid[(int)x*grid_p + (int)z] += b;

    } 
}



int main(int argc, char **argv)
{
    // arg[1] = number of rays, arg[2] = number of grid points
    if (argc != 3)
    {
        printf("Invalid number of arguments.\n");
        printf("To run: $ ./raytrace <number_of_rays> <number_of_gridpoints>\n");
        exit(1);
    }
    int n_rays = atoi(argv[1]);
    int grid_p = atoi(argv[2]);

    if (n_rays < (grid_p*grid_p))
    {
        printf("Invalid arguments: number of rays must be greater than grid_points^2\n");
        exit(1);
    }

	struct timeval start, end;
	gettimeofday(&start, NULL);

    srand(time(NULL));

    // allocate window (grid_p x grid_p) 
    double * grid = (double *) calloc(grid_p*grid_p, sizeof(double));  

	// copy data over launch kernel 

	// Cuda malloc
	double * cuda_grid;
	hipError_t _e;
	_e = hipMalloc((void**)&cuda_grid, grid_p*grid_p * sizeof(double));
	if (_e != hipSuccess)
		printf("Cuda error: %s\n", hipGetErrorString(_e));

	//transfer data to gpu
	_e = hipMemcpy(cuda_grid, grid, grid_p*grid_p*sizeof(double), hipMemcpyHostToDevice);
	if (_e != hipSuccess)
		printf("Cuda error: %s\n", hipGetErrorString(_e));

	// run kernel
	int block_size = ;
	int rays_per_thread = 50;
	int n_blocks = (n_rays + block_size - 1) / (block_size*rays_per_thread);
	printf("number of blocks = %d\n", n_blocks);

	raytrace<<< n_blocks, block_size>>>(cuda_grid, grid_p, rays_per_thread);
	_e = hipGetLastError();

	//get data from gpu
	_e = hipMemcpy(grid, cuda_grid, grid_p*grid_p*sizeof(double), hipMemcpyDeviceToHost);
	if (_e != hipSuccess)
		printf("Cuda error: %s\n", hipGetErrorString(_e));

    // print execution time
    gettimeofday(&end, NULL);
    double m = 1000000;
    double t = ((end.tv_sec*m + end.tv_usec) - (start.tv_sec*m + start.tv_usec));   
    printf("Cuda,%d, %g\n", n_rays, t / m);

    // write grid to file
    FILE * out = fopen("sphere.bin", "wb");

    for (int i = 0; i < grid_p; ++i)
        for (int j = 0; j < grid_p; ++j)
        {
            fwrite(&(grid[i*grid_p + j]), sizeof(double), 1, out);
        }            
    

    fclose(out);

    free(grid);

    return 0;
}

